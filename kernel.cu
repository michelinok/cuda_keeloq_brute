#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdint.h"
#include "time.h"
#include "stdlib.h"

#define KeeLoq_NLF		0x3A5C742E
#define bit(x,n)		(((x)>>(n))&1)
#define g5(x,a,b,c,d,e)	(bit(x,a)+bit(x,b)*2+bit(x,c)*4+bit(x,d)*8+bit(x,e)*16)

FILE* fp_log;

uint32_t* dev_ctext = nullptr;
uint32_t* dev_p01 = nullptr;
uint32_t* dev_p02 = nullptr;
uint32_t* dev_p11 = nullptr;
uint32_t* dev_p12 = nullptr;
uint32_t* dev_p21 = nullptr;
uint32_t* dev_p22 = nullptr;
uint32_t* dev_p31 = nullptr;
uint32_t* dev_p32 = nullptr;
uint64_t* dev_key0 = nullptr;
uint64_t* dev_key1 = nullptr;
uint64_t* dev_key2 = nullptr;
uint64_t* dev_key3 = nullptr;
uint64_t* dev_skey0 = nullptr;
uint64_t* dev_skey1 = nullptr;
uint64_t* dev_skey2 = nullptr;
uint64_t* dev_skey3 = nullptr;
uint32_t* dev_p1fin = nullptr;
uint32_t* dev_p2fin = nullptr;
uint32_t* dev_p3fin = nullptr;
uint64_t* dev_keyfin = nullptr;

__device__ uint32_t decrypt(const uint32_t data, const uint64_t key)
{
    uint32_t x = data, r;

    for (r = 0; r < 528; r++)
    {
        x = (x << 1) ^ bit(x, 31) ^ bit(x, 15) ^ (uint32_t)bit(key, (15 - r) & 63) ^ bit(KeeLoq_NLF, g5(x, 0, 8, 19, 25, 30));
    }
    return x;
}


__device__ __host__ uint64_t xorshift64(uint64_t x64)
{
    x64 ^= x64 << 13;
    x64 ^= x64 >> 7;
    x64 ^= x64 << 17;
    return x64;
}

__global__ void rekey(uint64_t* key0, uint64_t* key1, uint64_t* key2, uint64_t* key3, int size) {
    uint64_t val; 
    
    val = key0[0] = xorshift64(key3[size - 1]);
    val = key1[0] = xorshift64(val);
    val = key2[0] = xorshift64(val);
    val = key3[0] = xorshift64(val);


    for (int i = 1; i < size; i++) {
        val = key0[i] = xorshift64(val);
        val = key1[i] = xorshift64(val);
        val = key2[i] = xorshift64(val);
        val = key3[i] = xorshift64(val);
    }
 }

__global__ void Kernel(uint32_t* ctext, uint64_t* key, uint32_t* p1, uint32_t* p2, int size, uint64_t* finkey, uint32_t* finp1, uint32_t* finp2, uint32_t* finp3) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        p1[i] = decrypt(ctext[0], key[i]);
        p2[i] = decrypt(ctext[1], key[i]);

        if (p2[i] == (p1[i] + 1)) {
            finkey[0] = key[i];
            finp1[0] = p1[i];
            finp2[0] = p2[i];
            finp3[0] = decrypt(ctext[2], key[i]);
        }


    }
}


// Helper function for using CUDA to add vectors in parallel.
void initkey(uint64_t* key0, uint64_t* key1, uint64_t* key2, uint64_t* key3, uint32_t* ctext, int size) {

    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void**)&dev_key0, size * sizeof(uint64_t));
    hipMalloc((void**)&dev_key1, size * sizeof(uint64_t));
    hipMalloc((void**)&dev_key2, size * sizeof(uint64_t));
    hipMalloc((void**)&dev_key3, size * sizeof(uint64_t));
    //shaddowKeys
    hipMalloc((void**)&dev_skey0, size * sizeof(uint64_t));
    hipMalloc((void**)&dev_skey1, size * sizeof(uint64_t));
    hipMalloc((void**)&dev_skey2, size * sizeof(uint64_t));
    hipMalloc((void**)&dev_skey3, size * sizeof(uint64_t));

    hipMalloc((void**)&dev_p01, size * sizeof(uint32_t));
    hipMalloc((void**)&dev_p02, size * sizeof(uint32_t));
    hipMalloc((void**)&dev_p11, size * sizeof(uint32_t));
    hipMalloc((void**)&dev_p12, size * sizeof(uint32_t));
    hipMalloc((void**)&dev_p21, size * sizeof(uint32_t));
    hipMalloc((void**)&dev_p22, size * sizeof(uint32_t));
    hipMalloc((void**)&dev_p31, size * sizeof(uint32_t));
    hipMalloc((void**)&dev_p32, size * sizeof(uint32_t));

    hipMalloc((void**)&dev_keyfin, 2 * sizeof(uint64_t));
    hipMalloc((void**)&dev_p1fin, 2 * sizeof(uint32_t));
    hipMalloc((void**)&dev_p2fin, 2 * sizeof(uint32_t));
    hipMalloc((void**)&dev_p3fin, 2 * sizeof(uint32_t));
    hipMalloc((void**)&dev_ctext, 3 * sizeof(uint32_t));


    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_key0, key0, size * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_key1, key1, size * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_key2, key2, size * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_key3, key3, size * sizeof(uint64_t), hipMemcpyHostToDevice);

    hipMemcpy(dev_skey0, dev_key0, size * sizeof(uint64_t), hipMemcpyDeviceToDevice);
    hipMemcpy(dev_skey1, dev_key1, size * sizeof(uint64_t), hipMemcpyDeviceToDevice);
    hipMemcpy(dev_skey2, dev_key2, size * sizeof(uint64_t), hipMemcpyDeviceToDevice);
    hipMemcpy(dev_skey3, dev_key3, size * sizeof(uint64_t), hipMemcpyDeviceToDevice);

    hipMemcpy(dev_p01, 0, size * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_p02, 0, size * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_p11, 0, size * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_p12, 0, size * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_p21, 0, size * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_p22, 0, size * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_p31, 0, size * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_p32, 0, size * sizeof(uint32_t), hipMemcpyHostToDevice);

    hipMemcpy(dev_p1fin, 0, 2 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_p2fin, 0, 2 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_p3fin, 0, 2 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_keyfin, 0, 2 * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_ctext, ctext, 3 * sizeof(uint32_t), hipMemcpyHostToDevice);



    // Launch a kernel on the GPU with one thread for each element.
    // 2 is number of computational blocks and (size + 1) / 2 is a number of threads in a block
    //addKernel << <1, (size + 1) >> > (dev_key, dev_p1, dev_p2, size);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    //hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    //hipMemcpy(p1, dev_p1, size * sizeof(uint32_t), hipMemcpyDeviceToHost);
    //hipMemcpy(p2, dev_p2, size * sizeof(uint32_t), hipMemcpyDeviceToHost);

    //hipFree(dev_key);
    //hipFree(dev_p1);
    //hipFree(dev_p2);
}

int main(int argc, char** argv) {
    
    if (argc <= 3) {
        printf("Enter 3 Hopping-codes in Format 0x12345678 !\n");
        return - 1;

    }
    
    const int arraySize = 1024;
    //uint32_t p1[arraySize];
    //uint32_t p2[arraySize];
    uint64_t key0[arraySize];
    uint64_t key1[arraySize];
    uint64_t key2[arraySize];
    uint64_t key3[arraySize];
    uint64_t ctr = 0;
    uint64_t finalkey[2] = { 0 };
    uint32_t p1fin[2];
    uint32_t p2fin[2];
    uint32_t p3fin[2];

    uint32_t ctext[3];
    
    srand(time(NULL));


    sscanf(argv[1],"0x%08x", &ctext[0]);
    sscanf(argv[2], "0x%08x", &ctext[1]);
    sscanf(argv[3], "0x%08x", &ctext[2]);
    if (argc >= 5) {
        sscanf(argv[4], "0x%llx", &key0[0]);
    }

    if (key0[0] == 0) {
        key0[0] = xorshift64(xorshift64((time(NULL)))) * rand() + 0x1fffffffffffffffUl;
    }

    for (int i = 1; i < arraySize; i++) {
        key0[i] = xorshift64(key0[i - 1]);
    }
    key1[0] = xorshift64(key0[arraySize - 1]);

    for (int i = 1; i < arraySize; i++) {
        key1[i] = xorshift64(key1[i - 1]);
    }
    key2[0] = xorshift64(key1[arraySize - 1]);

    for (int i = 1; i < arraySize; i++) {
        key2[i] = xorshift64(key2[i - 1]);
    }
    key3[0] = xorshift64(key2[arraySize - 1]);

    for (int i = 1; i < arraySize; i++) {
        key3[i] = xorshift64(key3[i - 1]);
    }

    initkey(key0, key1, key2, key3, ctext, arraySize);

    ctext[0] = ctext[1] = ctext[2]= 0;

    hipMemcpy(ctext, dev_ctext, 3 * sizeof(uint32_t), hipMemcpyDeviceToHost);

    printf("Cuda accelerated Keeloq Bruteforcer...\n\nHoppingcodes: 0x%08X 0x%08X 0x%08X\nStartkey: 0x%llx\nPress Enter to Start Brute Force...\n", ctext[0], ctext[1], ctext[2],key0[0]);
    getchar();

    hipStream_t stream0,stream1,stream2,stream3,stream4;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);

    while (1 == 1) {
        
        Kernel << <2, (arraySize + 1) / 2, 0, stream0 >> > (dev_ctext, dev_key0, dev_p01, dev_p02, arraySize, dev_keyfin, dev_p1fin, dev_p2fin, dev_p3fin);
        Kernel << <2, (arraySize + 1) / 2, 0, stream1 >> > (dev_ctext, dev_key1, dev_p11, dev_p12, arraySize, dev_keyfin, dev_p1fin, dev_p2fin, dev_p3fin);
        Kernel << <2, (arraySize + 1) / 2, 0, stream2 >> > (dev_ctext, dev_key2, dev_p21, dev_p22, arraySize, dev_keyfin, dev_p1fin, dev_p2fin, dev_p3fin);
        Kernel << <2, (arraySize + 1) / 2, 0, stream3 >> > (dev_ctext, dev_key3, dev_p31, dev_p32, arraySize, dev_keyfin, dev_p1fin, dev_p2fin, dev_p3fin);
        rekey << <1, 1, 0, stream4 >> > (dev_skey0, dev_skey1, dev_skey2, dev_skey3, arraySize);
        hipDeviceSynchronize();

        hipMemcpy(dev_key0, dev_skey0, arraySize * sizeof(uint64_t), hipMemcpyDeviceToDevice);
        hipMemcpy(dev_key1, dev_skey1, arraySize * sizeof(uint64_t), hipMemcpyDeviceToDevice);
        hipMemcpy(dev_key2, dev_skey2, arraySize * sizeof(uint64_t), hipMemcpyDeviceToDevice);
        hipMemcpy(dev_key3, dev_skey3, arraySize * sizeof(uint64_t), hipMemcpyDeviceToDevice);

        hipMemcpy(finalkey, dev_keyfin, 2 * sizeof(uint64_t), hipMemcpyDeviceToHost);

        if (finalkey[0] != 0) {
            hipMemcpy(p1fin, dev_p1fin, 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
            hipMemcpy(p2fin, dev_p2fin, 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
            hipMemcpy(p3fin, dev_p3fin, 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
            
            if (p3fin[0] == (p2fin[0] + 1)) {
                fp_log = fopen("logfile.log", "a");
                fprintf(fp_log, "\nPossible Key Found!!! Key: %llX %04X / %04X / %04X Counter: %llX\n\a\a\a\a", finalkey[0], p1fin[0], p2fin[0], p3fin[0], ctr * arraySize * 4);
                printf("\nPossible Key Found!!! Key: %llX %04X / %04X / %04X Counter: %llX\n\a\a\a\a", finalkey[0], p1fin[0], p2fin[0], p3fin[0], ctr * arraySize * 4);
                fclose(fp_log);
                getchar();
                return 0;
            }
            else {
                fp_log = fopen("logfile.log", "a");
                fprintf(fp_log, "Match! Key: %llX %04X / %04X / %04X Counter: %llX\n", finalkey[0], p1fin[0], p2fin[0], p3fin[0], ctr* arraySize);
                printf("\nMatch! Key: %llX %04X / %04X / %04X Counter: %llX\n\a", finalkey[0], p1fin[0], p2fin[0], p3fin[0], ctr * arraySize);
                finalkey[0] = finalkey[1] = 0;
                fclose(fp_log);
                hipMemcpy(dev_keyfin, finalkey, 2 * sizeof(uint64_t), hipMemcpyHostToDevice);
            }
        }

        if (ctr % 0xFFFF == 0) {
            printf(">");
            //hipMemcpy(key, dev_key, arraySize * sizeof(uint64_t), hipMemcpyDeviceToHost);
            //hipMemcpy(p1, dev_p1, arraySize * sizeof(uint32_t), hipMemcpyDeviceToHost);
            //hipMemcpy(p2, dev_p2, arraySize * sizeof(uint32_t), hipMemcpyDeviceToHost);
            //printf("Key 0: %I64X %04X / %04X Counter: %I64X\n", key[0], p1[0], p2[0], ctr * arraySize);
        }

        //rekey << <1, 1 >> > (dev_key0, dev_key1, dev_key2, dev_key3, arraySize);
        ctr++;

    }

    hipDeviceReset();

    return 0;
}